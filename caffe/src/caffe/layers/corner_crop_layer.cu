#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/corner_crop_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void crop(const int num, const int channel, 
        const int height, const int width, 
        const int x, const int y, const int crop_h_, const int crop_w_,
        const Dtype* in_data, Dtype* out_data, const bool forward) {
  const int count = num*channel*height*width;
  int top_index;
  CUDA_KERNEL_LOOP(index, count) {
    const int w = index%width;
    const int h = (index/width)%height;
    const int c = (index/width/height)%channel;
    const int n = index/width/height/channel;
    if (y<=h && h<y+crop_h_ && x<=w && w<x+crop_w_) {
      top_index = ((n*channel+c)*crop_h_+h-y)*crop_w_+w-x;
      if (forward) {
         out_data[top_index] = in_data[index];
      } else {
        out_data[index] += in_data[top_index];
      }
    }
  }
}

template <typename Dtype>
__global__ void average(const int num, const int channel,
        const int height, const int width,
        Dtype* out_data, const int* count_crop) {
  const int count = num*channel*height*width;
  CUDA_KERNEL_LOOP(index, count) {
    const int w = index%width;
    const int h = (index/width)%height;
    if (count_crop[h*width+w]) {
      out_data[index] /= count_crop[h*width+w];
    }
  }
}

template <typename Dtype>
void CornerCropLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const int height = bottom[0]->height();
  const int width = bottom[0]->width();
  const int channel = bottom[0]->channels();
  const int num = bottom[0]->num();
  const int count = bottom[0]->count();
  int x, y;
  const int* points = points_.cpu_data();
  for (int i = 0; i < crop_num_; ++i) {
    Dtype* top_data = top[i]->mutable_gpu_data();
    x = points[i*2+0];
    y = points[i*2+1];
    crop<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        num, channel, height, width, x, y, crop_h_, crop_w_,
        bottom_data, top_data, true);
    CUDA_POST_KERNEL_CHECK;
  }
}

template <typename Dtype>
void CornerCropLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  caffe_gpu_set(bottom[0]->count(), Dtype(0), bottom_diff);
  const int height = bottom[0]->height();
  const int width = bottom[0]->width();
  const int channel = bottom[0]->channels();
  const int num = bottom[0]->num();
  const int count = bottom[0]->count();
  int x, y;
  const int* points = points_.cpu_data();
  for (int i = 0; i < crop_num_; ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    x = points[i*2+0];
    y = points[i*2+1];
    crop<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        num, channel, height, width, x, y, crop_h_, crop_w_,
        top_diff, bottom_diff, false);
    CUDA_POST_KERNEL_CHECK;
  }
  average<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          num, channel, height, width, bottom_diff, counts_.gpu_data());
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(CornerCropLayer);

}  // namespace caffe
